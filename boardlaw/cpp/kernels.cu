#include <math.h>
#include <hip/hip_math_constants.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

at::cuda::CUDAStream stream() { 
    return at::cuda::getCurrentCUDAStream();
}