#include <math.h>
#include <hip/hip_math_constants.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAException.h>

at::cuda::CUDAStream stream() { 
    return at::cuda::getCurrentCUDAStream();
}