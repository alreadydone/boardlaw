#include <math.h>
#include <hip/hip_math_constants.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

at::cuda::CUDAStream stream() { 
    return at::cuda::getCurrentCUDAStream();
}