#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <ATen/ATen.h>
#include "common.h"
#include <ATen/cuda/HIPContext.h>

at::cuda::CUDAStream stream() { 
    return at::cuda::getCurrentCUDAStream();
}

__global__ void solve_policy_kernel(
    TP2D::PTA pi, TP2D::PTA q, TP1D::PTA lambda_n,
    TP2D::PTA policy, TP1D::PTA alpha_star) {

    const auto B = pi.size(0);
    const auto A = pi.size(1);
    const int b = blockIdx.x;

    for (int a = 0; a < A; a++) {
        policy[b][a] = 1.f;
        alpha_star[b] = 1.f;
    }
}

__host__ Solution solve_policy(const TT pi, const TT q, const TT lambda_n) {
    const uint B = pi.size(0);
    const uint A = pi.size(1);

    Solution soln(B, A);

    solve_policy_kernel<<<{B}, {1}, 0, stream()>>>(
        TP2D(pi).pta(), TP2D(q).pta(), TP1D(lambda_n).pta(),
        TP2D(soln.policy).pta(), TP1D(soln.alpha_star).pta());

    return soln;
}