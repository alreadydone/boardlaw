#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_math_constants.h>
#include <ATen/ATen.h>
#include "common.h"
#include <ATen/cuda/HIPContext.h>

const uint BLOCK = 8;

using F1D = TensorProxy<float, 1>;
using F2D = TensorProxy<float, 2>;
using F3D = TensorProxy<float, 3>;
using I1D = TensorProxy<int, 1>;
using I2D = TensorProxy<int, 2>;
using I3D = TensorProxy<int, 3>;
using B1D = TensorProxy<bool, 1>;
using B2D = TensorProxy<bool, 2>;

at::cuda::CUDAStream stream() { 
    return at::cuda::getCurrentCUDAStream();
}

__global__ void solve_policy_kernel(
    F2D::PTA pi, F2D::PTA q, F1D::PTA lambda_n,
    F1D::PTA alpha_star) {

    const auto B = pi.size(0);
    const auto A = pi.size(1);
    const int b = blockIdx.x*blockDim.x + threadIdx.x;
    if (b >= B) {
        return;
    }

    // Copy data into shared memory
    extern __shared__ float shared[];
    float *qb = (float*)&shared[threadIdx.x*2*A];
    float *pib = (float*)&shared[threadIdx.x*2*A+A];
    for (int a = 0; a < A; a++) {
        qb[a] = q[b][a];
        pib[a] = pi[b][a];
    }
    __syncthreads();

    const auto lambda = lambda_n[b];

    // Find the initial alpha
    float alpha = 0.f;
    for (int a = 0; a < A; a++) {
        float gap = fmaxf(lambda*pib[a], 1.e-6f);
        alpha = fmaxf(alpha, qb[a] + gap);
    }

    float error = HIP_INF_F;
    float new_error = HIP_INF_F;
    // Typical problems converge in 10 steps. Hypothetically 100 might be 
    // hit sometimes, but it's worth risking it for how utterly awful it'd 
    // be debugging an infinite loop in the kernel.
    for (int s=0; s<100; s++) {
        float S = 0.f; 
        float g = 0.f;
        for (int a=0; a<A; a++) {
            float top = lambda*pib[a];
            float bot = alpha - qb[a];
            S += top/bot;
            g += -top/powf(bot, 2);
        }
        new_error = S - 1.f;
        // printf("%d: alpha: %.2f, S: %.2f, e: %.2f, g: %.2f\n", b, alpha, S, new_error, g);
        if ((new_error < 1e-3f) || (error == new_error)) {
            alpha_star[b] = alpha;
            break;
        } else {
            alpha -= new_error/g;
            error = new_error;
        }
    }
}

__host__ TT solve_policy(const TT pi, const TT q, const TT lambda_n) {
    const uint B = pi.size(0);
    const uint A = pi.size(1);

    F1D alpha_star(pi.new_empty({B}));
    alpha_star.t.fill_(NAN);

    //TODO: Replace this with a hardware dependent test
    assert (BLOCK*2*A*sizeof(float) < 64*1024);

    const uint n_blocks = (B + BLOCK - 1)/BLOCK;
    solve_policy_kernel<<<{n_blocks}, {BLOCK}, BLOCK*2*A*sizeof(float), stream()>>>(
        F2D(pi).pta(), F2D(q).pta(), F1D(lambda_n).pta(),
        alpha_star.pta());

    return alpha_star.t;
}

__global__ void descend_kernel(
    F3D::PTA logits, F3D::PTA w, I2D::PTA n, F1D::PTA c_puct,
    I2D::PTA seats, B2D::PTA terminal, I3D::PTA children,
    I1D::PTA parents, I1D::PTA actions) {

    const auto b = blockIdx.x;

    parents[b] = b;
    actions[b] = b;
}

__host__ DescentResult descend(
    const TT logits, const TT w, const TT n, const TT c_puct,
    const TT seats, const TT terminal, const TT children) {

    const uint B = logits.size(0);

    auto parents = seats.new_empty({B});
    auto actions = seats.new_empty({B});
    descend_kernel<<<{B}, {1}, 0, stream()>>>(
        F3D(logits).pta(), F3D(w).pta(), I2D(n).pta(), F1D(c_puct).pta(),
        I2D(seats).pta(), B2D(terminal).pta(), I3D(children).pta(),
        I1D(parents).pta(), I1D(actions).pta());

    return {parents, actions};
}